#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <vector>

#define VEC_DIM 10000
#define MAT_DIM1 10000
#define MAT_DIM2 10000
#define NUM_THREADS 100
#define NUM_BLOCKS 5

__global__ void custom_matmul(unsigned int *vec, unsigned int *mat,
                              unsigned int *result_ptr) {
  unsigned int result;
  int x = threadIdx.x;

  for (int j = 0; j < MAT_DIM1; j++) {
    result = 0;
    for (int i = 0; i < VEC_DIM; i += NUM_THREADS) {
      result += vec[i + x] * mat[j * MAT_DIM2 + i + x];
    }
    result_ptr[j] = result;
  }
}

int main() {

  std::cout << "start!\n";
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
  }

  hiprandGenerator_t generator;
  hiprandRngType_t rng_type = hiprandRngType_t::HIPRAND_RNG_PSEUDO_DEFAULT;

  void *mem_ptr = 0; // allocated memory
  hipMalloc(&mem_ptr, sizeof(int) * VEC_DIM);

  hiprandCreateGenerator(&generator, rng_type);

  unsigned int *vec_ptr = reinterpret_cast<unsigned int *>(mem_ptr);
  hiprandGenerate(generator, vec_ptr, VEC_DIM);

  void *mem_ptr2 = 0; // allocated memory
  hipMalloc(&mem_ptr2, sizeof(int) * MAT_DIM1 * MAT_DIM2);

  unsigned int *mat_ptr = reinterpret_cast<unsigned int *>(mem_ptr2);
  hiprandGenerate(generator, mat_ptr, MAT_DIM1 * MAT_DIM2);

  unsigned int *result_ptr = 0;
  hipMalloc(&result_ptr, sizeof(int) * VEC_DIM);

  custom_matmul<<<1, NUM_THREADS>>>(vec_ptr, mat_ptr, result_ptr);
  hipDeviceSynchronize();

  // test code
  unsigned int *host_ptr =
      reinterpret_cast<unsigned int *>(malloc(sizeof(int) * VEC_DIM));
  hipMemcpy(host_ptr, result_ptr, sizeof(int) * 200, hipMemcpyDeviceToHost);
  for (int i = 0; i < 10; i++) {
    std::cout << host_ptr[i] << std::endl;
  }
  free(host_ptr);

  hiprandDestroyGenerator(generator);
  hipFree(vec_ptr);
  hipFree(mat_ptr);
  hipFree(result_ptr);

  std::cout << "finish!";
}